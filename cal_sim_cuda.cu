//calculate cosine similarity from a cell to all of the other cells parallelized by CUDA

#include <hip/hip_runtime.h>
#include<iostream>
#include<iomanip>
#include<vector>
#include<fstream>
#include<sstream>
#include <cstdlib>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include<time.h>

using namespace std;
typedef vector<vector<float> > Matrix;
//read a scRNA-seq expression table into a matrix

__global__ void cal_dis(float *train_data, float *test_data, int startlabel, int D, int N, float *dis,int pitch)
{//D is the dimension(#genes) of vector(cell), N is #cell
	int tid = blockIdx.x;
	if(tid<startlabel){
		dis[tid]=0;
	}else if(tid<N){
		float traintemp = 0;
		float testtemp =0;
		float dottemp =0;
		float trainsum = 0;
		float testsum = 0;
		float dotsum = 0;
		
		for(int i=0;i<D;i++){
			traintemp = *((float*)((char*)train_data + tid * pitch) + i);
			testtemp =test_data[i];
			dottemp = *((float*)((char*)train_data + tid * pitch) + i) * test_data[i];
			dotsum += dottemp;
			trainsum += traintemp*traintemp;
			testsum += testtemp*testtemp;
		}
		dis[tid] = dotsum/(sqrt(testsum)*sqrt(trainsum));
	}
}

void print(float *data,int start_i, int n)
{
    for(int i=start_i;i<n;i++)
    {
        cout<<start_i<<"\t"<<i<<"\t"<<data[i]<<"\n";
    }
    //cout<<endl;
}


int main(int argc, const char** argv){
	int the_cell=0;
	if(argc!=3){
		cout<<"\nCalculate cosine simialrity scores between a cell and each of the others\n******\nUSAGE:\n******\n";
		cout<<argv[0]<<" <expression_file> <this cell>  > OutputFile\n";
		cout<<endl;
		exit(1);
	}
	ifstream express_file(argv[1]);

	string scell(argv[2]);
	istringstream iscell(scell);
	iscell>>the_cell;// current cell label

	string firstline; getline(express_file,firstline);//read the first line
	Matrix ExpressMatrix; //define a matrix to store the expression values
	string GeneName;
	vector<string> allGeneNames;
	for(string s;getline(express_file,s);){//read the input file into a matrix
		vector<float> thisline;
		istringstream sin(s);
		sin>>GeneName;
		allGeneNames.push_back(GeneName);
		for(float a; sin>>a; ){
			thisline.push_back(a);
		}
		ExpressMatrix.push_back(thisline);
		thisline.clear();
	}
	int cell_num=ExpressMatrix[0].size();
	int gene_num=ExpressMatrix.size();
	//int gene_num=allGeneNames.size();
	//cout<<"#cells="<<cell_num<<"; #genes="<<gene_num<<endl;
	
	float *h_train_data, *h_test_data ;
	
	h_train_data=(float *)malloc(gene_num*cell_num*sizeof(float));
	h_test_data=(float *)malloc(gene_num*sizeof(float));
	
	for(int i=0;i<cell_num;i++){//transpose of the Expression Matrix and normalize
		/*float thiscellsum=0;
		for(int j=0;j<gene_num;j++){
			thiscellsum+=ExpressMatrix[j][i];
		}*/
		for(int j=0;j<gene_num;j++){
			h_train_data[i*gene_num+j]=ExpressMatrix[j][i];
			//h_train_data[i][j]=log(((ExpressMatrix[j][i])/thiscellsum)*10000+1);//normalize
		}
	}
	int N=cell_num;
	float distance[N];
	float *d_train_data , *d_test_data , *d_dis;

	size_t pitch_d;
	size_t pitch_h = gene_num * sizeof(int) ;


	//allocate memory on GPU 
	hipMallocPitch( &d_train_data , &pitch_d , gene_num * sizeof(float) , N ); 
	hipMalloc( (void**)&d_test_data ,  gene_num*sizeof(float) );
	hipMalloc( (void**)&d_dis , N*sizeof(float) );
		

	for(int j=0;j<gene_num;j++){
		h_test_data[j]=h_train_data[the_cell*gene_num+j];//
	}
	
		
	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data , pitch_d , h_train_data , pitch_h , gene_num * sizeof(float) , N , hipMemcpyHostToDevice );
	hipMemcpy( d_test_data,  h_test_data ,  gene_num*sizeof(float), hipMemcpyHostToDevice);

	//calculate the distance
	cal_dis<<<N,1>>>( d_train_data,d_test_data,the_cell,gene_num,N,d_dis,pitch_d );

	//copy distance data from device to host
	hipMemcpy( distance , d_dis  , N*sizeof(float) , hipMemcpyDeviceToHost);

	//cout<<"distance:"<<endl;;
	print(distance , the_cell, cell_num);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	
	return 0;
}
