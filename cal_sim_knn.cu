//calculate cosine similarity from a cell to all of the other cells parallelized by CUDA
#include<iostream>
#include<iomanip>
#include<vector>
#include<fstream>
#include<sstream>
#include<map>
#include<cmath>
#include<algorithm>
#include<numeric>
#include <cstdlib>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include<time.h>
#include <hip/hip_runtime.h>

using namespace std;
typedef vector<vector<float> > Matrix;
//read a scRNA-seq expression table into a matrix
typedef pair<int, float> intfloatPAIR;

struct CmpByValue {
	bool operator()(const intfloatPAIR& lhs, const intfloatPAIR& rhs) {
		return lhs.second > rhs.second;
	}
};



__global__ void cal_dis(float *train_data, float *test_data, int D, int N, float *dis,int pitch)
{//D is the dimension(#genes) of vector(cell), N is #cell
	//int tid = blockIdx.x;
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid<N){
		float traintemp = 0;
		float testtemp =0;
		float dottemp =0;
		float trainsum = 0;
		float testsum = 0;
		float dotsum = 0;
		
		for(int i=0;i<D;i++){
			traintemp = *((float*)((char*)train_data + tid * pitch) + i);
			testtemp =test_data[i];
			dottemp = *((float*)((char*)train_data + tid * pitch) + i) * test_data[i];
			dotsum += dottemp;
			trainsum += traintemp*traintemp;
			testsum += testtemp*testtemp;
		}
		dis[tid] = dotsum/(sqrt(testsum)*sqrt(trainsum));
	}
}

void print(float *data,int start_i, int n, float cutoff)
{
   for(int i=start_i+1;i<n;i++)
    {
		if(data[i]>=cutoff){
			cout<<start_i<<"\t"<<i<<"\t"<<data[i]<<"\n";
		}
    }
}

void knn(float *data,int start_i, int n, int k)
{//k-nearest neighbors of start_i
    vector<intfloatPAIR> i2fVector;
	i2fVector.clear();
	for(int i=0;i<n;i++)
    {
        if(i!=start_i){
			intfloatPAIR ifp=make_pair(i,data[i]);
			if(i2fVector.size()<k){
				i2fVector.push_back(ifp);
			}else{
				sort(i2fVector.begin(), i2fVector.end(), CmpByValue());//sort second in descending order
				if(i2fVector[i2fVector.size()-1].second<data[i]){//if the last one less than data[i],replace it by data[i]
					i2fVector.pop_back(); //delete the min(first) one
					i2fVector.push_back(ifp);
				}
			}
		}
    }
	for (int i=0;i<i2fVector.size();i++)
	{//print the KNNs of start_i
		cout<<start_i<<"\t"<<i2fVector[i].first<<"\t"<<i2fVector[i].second<<"\n";
	}
}





int main(int argc, const char** argv){
	int the_cell=0;
	int K=10;
	float cutoff=0.8;
	if(argc!=5){
		cout<<"\nCalculate cosine simialrity scores between a cell and each of the others\n******\nUSAGE:\n******\n";
		cout<<argv[0]<<" <expression_file> <this cell>  <K>  <cutoff>  > OutputFile\n";
		cout<<"\n Note: <this cell> is order of a cell, e.g. 0,1,2,... \n";
		cout<<"K is an integral number (K-nearest neighbors). e.g. K=5\n";
		cout<<"cutoff: is the normalized cell-cell simialrity score between 0 and 1. e.g. cutoff=0.95\n"; 
		cout<<endl;
		exit(1);
	}
	ifstream express_file(argv[1]);

	string scell(argv[2]);
	istringstream iscell(scell);
	iscell>>the_cell;// current cell label
	
	string sk(argv[3]);
	istringstream isk(sk);
	isk>>K;// K of KNN
	
	string scut(argv[4]);
	istringstream iscut(scut);
	iscut>>cutoff;

	string firstline; getline(express_file,firstline);//read the first line
	Matrix ExpressMatrix; //define a matrix to store the expression values
	string GeneName;
	vector<string> allGeneNames;
	for(string s;getline(express_file,s);){//read the input file into a matrix
		vector<float> thisline;
		istringstream sin(s);
		sin>>GeneName;
		allGeneNames.push_back(GeneName);
		for(float a; sin>>a; ){
			thisline.push_back(a);
		}
		ExpressMatrix.push_back(thisline);
		thisline.clear();
	}
	int cell_num=ExpressMatrix[0].size();
	int gene_num=ExpressMatrix.size();
	//int gene_num=allGeneNames.size();
	//cout<<"#cells="<<cell_num<<"; #genes="<<gene_num<<endl;
	
	float *h_train_data, *h_test_data ;
	
	h_train_data=(float *)malloc(gene_num*cell_num*sizeof(float));
	h_test_data=(float *)malloc(gene_num*sizeof(float));
	
	for(int i=0;i<cell_num;i++){//transpose of the Expression Matrix and normalize
		/*float thiscellsum=0;
		for(int j=0;j<gene_num;j++){
			thiscellsum+=ExpressMatrix[j][i];
		}*/
		for(int j=0;j<gene_num;j++){
			h_train_data[i*gene_num+j]=ExpressMatrix[j][i];
			//h_train_data[i][j]=log(((ExpressMatrix[j][i])/thiscellsum)*10000+1);//normalize
		}
	}
	int N=cell_num;
	float distance[N];
	float *d_train_data , *d_test_data , *d_dis;

	size_t pitch_d;
	size_t pitch_h = gene_num * sizeof(int) ;


	//allocate memory on GPU 
	hipMallocPitch( &d_train_data , &pitch_d , gene_num * sizeof(float) , N ); 
	hipMalloc( (void**)&d_test_data ,  gene_num*sizeof(float) );
	hipMalloc( (void**)&d_dis , N*sizeof(float) );
		

	for(int j=0;j<gene_num;j++){
		h_test_data[j]=h_train_data[the_cell*gene_num+j];//
	}
	
		
	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data , pitch_d , h_train_data , pitch_h , gene_num * sizeof(float) , N , hipMemcpyHostToDevice );
	hipMemcpy( d_test_data,  h_test_data ,  gene_num*sizeof(float), hipMemcpyHostToDevice);

	int threadPerBlock =64;
	int blocksPerGrid=(N+ threadPerBlock-1)/threadPerBlock;


	//calculate the distance
	cal_dis<<<blocksPerGrid, threadPerBlock>>>( d_train_data,d_test_data,gene_num,N,d_dis,pitch_d );

	//copy distance data from device to host
	hipMemcpy( distance , d_dis  , N*sizeof(float) , hipMemcpyDeviceToHost);

	//cout<<"distance:"<<endl;;
	print(distance , the_cell, cell_num, cutoff);
	
	knn(distance, the_cell,cell_num, K);
	
	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	
	return 0;
}
